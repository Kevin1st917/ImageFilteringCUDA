// nvcc fft_cuda_2d.cu -lcublas -lcufft -arch=compute_52 -o fft_cuda_2d
//https://www.researchgate.net/figure/Computing-2D-FFT-of-size-NX-NY-using-CUDAs-cuFFT-library-49-FFT-fast-Fourier_fig3_324060154
#include "hip/hip_runtime.h"


#include <hipfft/hipfft.h>
 
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <vector>

#define DIM 4*65536//65536 = 256 * 256
#define NX 220
#define NY 220
using namespace std;



int main()
{
    int n = 0; //n is the number of the integers in the file ==> 12
    int data[220*220];
    int x;

    ifstream File;
    File.open("lenna_grayscale.txt");
    if(!File.is_open()){
        cout<<"It failed"<<endl;
        return 0;
    }

    while(File>>x){
        data[n] = x; 
        n++;
    }

    File.close();
    cout<<"n : "<<n<<endl;
    for(int i=0;i<n;i++){
        cout << data[i] << " ";
    }




    float elapsedTime = 0;
    hipfftHandle plan;
    hipfftComplex *host_data = (hipfftComplex*)malloc(NX*NY*sizeof(hipfftComplex));
    hipfftComplex *fft_data = (hipfftComplex*)malloc(NX*NY*sizeof(hipfftComplex));
    hipfftComplex *dev_data;
    hipEvent_t start,stop;
    
    //FEED INPUT
    srand(time(NULL));
    for(int i = 0;i<NX;i++){
        for(int j = 0;j<NY;j++){
            host_data[i*NY+j].x = (float)data[i*NY+j];  //rand()/(float)RAND_MAX;
            host_data[i*NY+j].y = 0.0;        
        }
    }

    //SHOW HOST DATA
    for(int i = 0;i<16;i++){
        printf("DATA: %3.1f %3.1f \n",host_data[i*NY+1].x,host_data[i*NY+1].y);
    }

    //ALLOCATE GPU MEMORY
    hipMalloc((void**)&dev_data,sizeof(hipfftComplex)*NX*NY);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    
    //COPY INPUT
    hipMemcpy(dev_data,host_data,NX*NY*sizeof(hipfftComplex),hipMemcpyHostToDevice);
    
    //CREATE CUFFT PLAN
    hipfftPlan2d(&plan,NX,NY,HIPFFT_C2C);
    
    //PERFORM COMPUTATION(fft and ifft)
    hipfftExecC2C(plan,dev_data,dev_data,HIPFFT_FORWARD);
    //COPY BACK RESULTS
    hipMemcpy(fft_data,dev_data,sizeof(hipfftComplex)*NX*NY,hipMemcpyDeviceToHost);
    ofstream outfile2;
    outfile2.open("fft_data.txt");
    // int data2[220*220] = {0};
    for(int i = 0;i<NX;i++){
        for(int j = 0;j<NY;j++){
            if(j == NY - 1){
                outfile2<<fft_data[i*NY+j].x<<endl;
            }else{
                outfile2<<fft_data[i*NY+j].x<<","; 
            }     
        }
    }
    outfile2.close(); 
    


    hipfftExecC2C(plan,dev_data,dev_data,HIPFFT_BACKWARD);//https://stackoverflow.com/questions/46562575/how-to-cuda-ifft
    
    //COPY BACK RESULTS
    hipMemcpy(host_data,dev_data,sizeof(hipfftComplex)*NX*NY,hipMemcpyDeviceToHost);
    
    //GET CALCULATION TIME
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    
    //SHOW RESULTS
    for(int i = 0;i<16;i++){
        printf("DATA: %3.1f %3.1f \n",host_data[i*NY+1].x/(NX*NY),host_data[i*NY+1].y/(NX*NY));
    }
    ofstream outfile;
    outfile.open("output_data.txt");
    // int data2[220*220] = {0};
    for(int i = 0;i<NX;i++){
        for(int j = 0;j<NY;j++){
            // data2[i*NY+j] = host_data[i*NY+3].x/(NX*NY)
            if(j == NY - 1){
                outfile<<host_data[i*NY+j].x/(NX*NY)<<endl;
            }else{
                outfile<<host_data[i*NY+j].x/(NX*NY)<<","; 
            }
              
        }
    }
    outfile.close();
    //FREEE MEMORY
    hipfftDestroy(plan);
    hipFree(dev_data);
    free(host_data);
    printf("elapsed time %f\n",elapsedTime);
    printf("CUFFT Calculation COMPLETED IN : % 5.3f ms \n",elapsedTime);  
}
