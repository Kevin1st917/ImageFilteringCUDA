// nvcc fft_cuda_2d.cu -lcublas -lcufft -arch=compute_52 -o fft_cuda_2d
//https://www.researchgate.net/figure/Computing-2D-FFT-of-size-NX-NY-using-CUDAs-cuFFT-library-49-FFT-fast-Fourier_fig3_324060154
#include "hip/hip_runtime.h"


#include <hipfft/hipfft.h>
 
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
 
#define DIM 4*65536//65536 = 256 * 256
#define NX 256
#define NY 256

int main()
{
    float elapsedTime = 0;
    hipfftHandle plan;
    hipfftComplex *host_data = (hipfftComplex*)malloc(NX*NY*sizeof(hipfftComplex));
    hipfftComplex *dev_data;
    hipEvent_t start,stop;
    
    //FEED INPUT
    srand(time(NULL));
    for(int i = 0;i<NX;i++){
        for(int j = 0;j<NY;j++){
            host_data[i*NY+j].x = (float)i;  //rand()/(float)RAND_MAX;
            host_data[i*NY+j].y = 0.0;        
        }
    }

    //SHOW HOST DATA
    for(int i = 0;i<16;i++){
        printf("DATA: %3.1f %3.1f \n",host_data[i*NY+3].x,host_data[i*NY+3].y);
    }

    //ALLOCATE GPU MEMORY
    hipMalloc((void**)&dev_data,sizeof(hipfftComplex)*NX*NY);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    
    //COPY INPUT
    hipMemcpy(dev_data,host_data,NX*NY*sizeof(hipfftComplex),hipMemcpyHostToDevice);
    
    //CREATE CUFFT PLAN
    hipfftPlan2d(&plan,NX,NY,HIPFFT_C2C);
    
    //PERFORM COMPUTATION(fft and ifft)
    hipfftExecC2C(plan,dev_data,dev_data,HIPFFT_FORWARD);

    hipfftExecC2C(plan,dev_data,dev_data,HIPFFT_BACKWARD);//https://stackoverflow.com/questions/46562575/how-to-cuda-ifft
    
    //COPY BACK RESULTS
    hipMemcpy(host_data,dev_data,sizeof(hipfftComplex)*NX*NY,hipMemcpyDeviceToHost);
    
    //GET CALCULATION TIME
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    
    //SHOW RESULTS
    for(int i = 0;i<16;i++){
        printf("DATA: %3.1f %3.1f \n",host_data[i*NY+3].x/(NX*NY),host_data[i*NY+3].y/(NX*NY));
    }
    //FREEE MEMORY
    hipfftDestroy(plan);
    hipFree(dev_data);
    free(host_data);
    printf("elapsed time %f\n",elapsedTime);
    printf("CUFFT Calculation COMPLETED IN : % 5.3f ms \n",elapsedTime);
}
