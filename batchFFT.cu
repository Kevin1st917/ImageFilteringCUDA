/*
 * The FFT is a divide-and-conquer algorithm for efficiently computing 
 * discrete Fourier transforms of complex or real-valued data sets.
 * 
 */

#include "header.cuh"

// CUDA ERROR CHECK
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/*
 * Should include <hipfft/hipfft.h>
 * C2C - Complex input to complex output
 * R2C - Real input to complex output
 * C2R - Symmetric complex input to real output
 *
 * These batched transforms have higher performance than single transforms.
 * In-place and out-of-place transforms
 * Execution of transforms across multiple GPUs
 * Streamed execution, enabling asynchronous computation and data movement
 */

int batchFFT(hipfftComplex* inputData, hipfftComplex* outputData,int BATCH, int DATASIZE)
{

	// --- Device side input data allocation and initialization
	hipfftComplex *deviceInputData; 
	gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftComplex)));
	hipMemcpy(deviceInputData, inputData, DATASIZE * BATCH * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	// --- Batched 1D FFTs
	hipfftHandle handle;
	int rank = 1;                           // --- 1D FFTs
	int n[] = { DATASIZE };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = DATASIZE, odist = DATASIZE; // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = BATCH;                      // --- Number of batched executions
	//Creates a plan supporting batched input and strided data layouts
	hipfftPlanMany(&handle, rank, n,
		inembed, istride, idist,
		onembed, ostride, odist, HIPFFT_C2C, batch);

	//hipfftPlan1d(&handle, DATASIZE, HIPFFT_R2C, BATCH);
	hipfftExecC2C(handle, deviceInputData, outputData, HIPFFT_FORWARD);
	hipfftDestroy(handle);
	

	return 0;
}